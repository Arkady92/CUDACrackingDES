#include "hip/hip_runtime.h"
#include "Arrays.cuh"
#include "CrackingDES.cuh"

inline void gpuAssert(hipError_t code, char *file, int line)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}

struct result
{
	bool isCracked;
	int keyNumber;
};

void encipherTextCPU(short * message, short * key, short * cipherMessage)
{
	short C[SHIFTSLEN+1][BLOCKSLEN];
	short D[SHIFTSLEN+1][BLOCKSLEN];
	short L[IPMSGCOUNT+1][MSGBITLEN/2];
	short R[IPMSGCOUNT+1][MSGBITLEN/2];
	short expandedR[EXTENDEDLEN];
	short sboxes[SBOXCOUNT][SBOXSIZE];
	short keys[KEYCOUNT][PC2LEN];

	for(int i = 0; i < BLOCKSLEN; i++)
	{
		C[0][i] = key[PC1[i]-1];
		D[0][i] = key[PC1[BLOCKSLEN + i]-1];
	}
	for(int i = 1; i < SHIFTSLEN+1; i++)
	{
		for(int j = 0; j < BLOCKSLEN - leftShifts[i]; j++)
		{
			C[i][j] = C[i-1][j + leftShifts[i]];
			D[i][j] = D[i-1][j + leftShifts[i]];
		}
		for(int j = 0; j < leftShifts[i]; j++)
		{
			C[i][j + BLOCKSLEN - leftShifts[i]] = C[i-1][j];
			D[i][j + BLOCKSLEN - leftShifts[i]] = D[i-1][j];
		}
		for(int j = 0; j < PC2LEN; j++)
		{
			if(PC2[j] - 1 < BLOCKSLEN)
				keys[i-1][j] = C[i][PC2[j]-1];
			else
				keys[i-1][j] = D[i][PC2[j]-BLOCKSLEN-1];
		}
	}

	for(int i = 0; i < MSGBITLEN/2; i++)
	{
		L[0][i] = message[IP[i]-1];
		R[0][i] = message[IP[MSGBITLEN/2 + i]-1];
	}

	for(int i = 1; i < IPMSGCOUNT+1; i++)
	{
		for(int j = 0; j < EXTENDEDLEN; j++)
			expandedR[j] = R[i-1][selectionTable[j] - 1] ^ keys[i-1][j];
		for(int j = 0; j < SBOXCOUNT; j++)
		{
			short row = 2 * expandedR[j*SBLOCKSIZE] + expandedR[j*SBLOCKSIZE + 5];
			short column = 8 * expandedR[j*SBLOCKSIZE + 1] 
			+ 4 * expandedR[j*SBLOCKSIZE + 2] + 2 * expandedR[j*SBLOCKSIZE + 3]
			+ expandedR[j*SBLOCKSIZE + 4];
			short sValue = S[j][row*SCOLUMNS + column];
			short mask = 1;
			for(int k = 0; k < SBOXSIZE; k++)
				sboxes[j][SBOXSIZE - k -1] = (sValue & (mask << k)) >> k;
		}
					for(int j = 0; j < MSGBITLEN/2; j++)
		{
			L[i][j] = R[i-1][j];
			R[i][j] = (L[i-1][j] + sboxes[(P[j]-1) / SBOXSIZE][(P[j]-1) % SBOXSIZE]) % 2;
		}
	}
	for(int i = 0; i < MSGBITLEN; i++)
	{
		if(reverseIP[i] < MSGBITLEN/2)
			cipherMessage[i] = R[16][reverseIP[i] - 1];
		else
			cipherMessage[i] = L[16][reverseIP[i] - 1 - MSGBITLEN/2];
	}
}

__device__ void encipherTextGPU(short * message, short * key, short * cipherMessage, bool * result)
{
	short C[SHIFTSLEN+1][BLOCKSLEN];
	short D[SHIFTSLEN+1][BLOCKSLEN];
	short L[IPMSGCOUNT+1][MSGBITLEN/2];
	short R[IPMSGCOUNT+1][MSGBITLEN/2];
	short expandedR[EXTENDEDLEN];
	short sboxes[SBOXCOUNT][SBOXSIZE];
	short keys[KEYCOUNT][PC2LEN];

	for(int i = 0; i < BLOCKSLEN; i++)
	{
		C[0][i] = key[d_PC1[i]-1];
		D[0][i] = key[d_PC1[BLOCKSLEN + i]-1];
	}
	for(int i = 1; i < SHIFTSLEN+1; i++)
	{
		for(int j = 0; j < BLOCKSLEN - d_leftShifts[i]; j++)
		{
			C[i][j] = C[i-1][j + d_leftShifts[i]];
			D[i][j] = D[i-1][j + d_leftShifts[i]];
		}
		for(int j = 0; j < d_leftShifts[i]; j++)
		{
			C[i][j + BLOCKSLEN - d_leftShifts[i]] = C[i-1][j];
			D[i][j + BLOCKSLEN - d_leftShifts[i]] = D[i-1][j];
		}
		for(int j = 0; j < PC2LEN; j++)
		{
			if(d_PC2[j] - 1 < BLOCKSLEN)
				keys[i-1][j] = C[i][d_PC2[j]-1];
			else
				keys[i-1][j] = D[i][d_PC2[j]-BLOCKSLEN-1];
		}
	}
	for(int i = 0; i < MSGBITLEN/2; i++)
	{
		L[0][i] = message[d_IP[i]-1];
		R[0][i] = message[d_IP[MSGBITLEN/2 + i]-1];
	}
	for(int i = 1; i < IPMSGCOUNT+1; i++)
	{
		for(int j = 0; j < EXTENDEDLEN; j++)
			expandedR[j] = R[i-1][d_selectionTable[j] - 1] ^ keys[i-1][j];
		for(int j = 0; j < SBOXCOUNT; j++)
		{
			short row = 2 * expandedR[j*SBLOCKSIZE] + expandedR[j*SBLOCKSIZE + 5];
			short column = 8 * expandedR[j*SBLOCKSIZE + 1] 
			+ 4 * expandedR[j*SBLOCKSIZE + 2] + 2 * expandedR[j*SBLOCKSIZE + 3]
			+ expandedR[j*SBLOCKSIZE + 4];
			short sValue = d_S[j][row*SCOLUMNS + column];
			short mask = 1;
		for(int k = 0; k < SBOXSIZE; k++)
				sboxes[j][SBOXSIZE - k -1] = (sValue & (mask << k)) >> k;
		}

		for(int j = 0; j < MSGBITLEN/2; j++)
		{
			L[i][j] = R[i-1][j];
			R[i][j] = (L[i-1][j] + sboxes[(d_P[j]-1) / SBOXSIZE][(d_P[j]-1) % SBOXSIZE]) % 2;
		}
	}
	*result = true;
	for(int i = 0; i < MSGBITLEN; i++)
	{
		if(d_reverseIP[i] < MSGBITLEN/2)
		{
			if(R[16][d_reverseIP[i] - 1] != cipherMessage[i])
			{
				*result = false;
				break;
			}
		}
		else if(L[16][d_reverseIP[i] - 1 - MSGBITLEN/2] != cipherMessage[i])
		{
			*result = false;
			break;
		}
	}
	if(*result)
		return;
}

__host__ __device__ void convertSignToBitArray(char sign, short * resultArray)
{
	//memset(resultArray, 0 ,SIGN_SIZE);
	char mask = 1;
	for(int i = 0; i < SIGN_SIZE; i++)
	 resultArray[i] = (sign & (mask << i)) >> i;
}

__host__ __device__ void convertTextToBitArray(char * text, int length, short * resultArray)
{
	//memset(resultArray, 0 ,length);
	for(int i = 0; i < MAX_TEXT_LEN; i++)
	{
		if(i < length)
			convertSignToBitArray(text[i],resultArray + i*SIGN_SIZE);
		else
			convertSignToBitArray('a',resultArray + i*SIGN_SIZE);
	}
}

void generateRandomPermutation(int signsCount, int length, char *resultArray)
{
	for(int i = 0; i < length; i++)
		resultArray[i] = 'a' + rand() % signsCount;
}

__host__ __device__ void generatePermutation(unsigned long long combination, int signsCount, int length, char * resultArray)
{
	for(int i = 0; i < length; i++)
	{
		int res = combination % signsCount;
		resultArray[i] = 'a' + res;
		combination /= signsCount;
	}
}

__global__ void CrackingDESKernel(short * _cipherText, short * _plainText, int signsCount, unsigned long long threadsCount, int group, int keyLength, struct result * result)
{
	__shared__ short cipherText[MSGBITLEN];
	__shared__ short plainText[MSGBITLEN];
	
	unsigned long long position = (blockIdx.x + group * MAXBLOCKCOUNT) * BLOCKSIZE + threadIdx.x;

	if(threadIdx.x < MSGBITLEN)
	{
		cipherText[threadIdx.x] = _cipherText[threadIdx.x];
		plainText[threadIdx.x] = _plainText[threadIdx.x];
	}
	__syncthreads();

	if(position >= threadsCount)
		return;
	char * code = new char[MSGLEN];
	short * key = new short[MSGBITLEN];
	bool * res = new bool[1];
	generatePermutation(position, signsCount, MSGLEN, code);
	convertTextToBitArray(code,keyLength,key);
	encipherTextGPU(plainText, key, cipherText, res);
	if(*res)
	{
		result->isCracked = true;
		result->keyNumber = position;
	}
	delete[] code;
	delete[] key;
	delete[] res;

	return;
}
void ERR(char *msg)
{
	fprintf(stderr,"Error: %s\n", msg);
	exit(1);
}

int main()
{
	char * plainText = new char[MSGLEN+1];
	char * key = new char[MSGLEN+1];
	short * plainBitText = new short[MSGBITLEN];
	short * cipherBitText = new short[MSGBITLEN];
	short * keyBit = new short[MSGBITLEN];
	hipEvent_t timerStart, timerStop;
	float timer;

	short * d_cipherText, * d_plainText;

	int signsCount = 0;
	printf("Enter the alphabet size (from 1 to 26).\n");
	scanf("%d", &signsCount);

	printf("Enter the plain text (maximum 8 signs).\n");
	scanf("%s", plainText);
	convertTextToBitArray(plainText,8,plainBitText);

	printf("Enter the key text (maximum 8 signs).\n");
	scanf("%s", key); 
	int keyLength = strlen(key);

	int option = 0;
	printf("Choose cracking type: 0 - sequentialy, 1 - randomize.\n");
	scanf("%d", &option);

	convertTextToBitArray(key,keyLength,keyBit);

	encipherTextCPU(plainBitText, keyBit, cipherBitText);

	printf("Cipher text generated from given text and key, now lets try to crack it.\n");

	if(hipMalloc((void**) &d_cipherText, sizeof(short)*MSGBITLEN) != hipSuccess)
		ERR("hipMalloc");
	if(hipMemcpy(d_cipherText, cipherBitText, sizeof(short)*MSGBITLEN, hipMemcpyHostToDevice) != hipSuccess)
		ERR("hipMemcpy");
	if(hipMalloc((void**) &d_plainText, sizeof(short)*MSGBITLEN) != hipSuccess)
		ERR("hipMalloc");

	char * code = new char[MSGLEN];
	struct result * result = new struct result;
	result->isCracked = false;
	result->keyNumber = -1;
	struct result * d_result;
	if(hipMalloc((void**) &d_result, sizeof(struct result)) != hipSuccess)
		ERR("hipMalloc");
	if(hipMemcpy(d_result, result, sizeof(struct result), hipMemcpyHostToDevice) != hipSuccess)
		ERR("hipMemcpy");

	unsigned long long threadsCount = 1;
	for(int i = 0; i < keyLength; i++)
		threadsCount *= signsCount;
	int blocksCount = threadsCount / BLOCKSIZE + 1;
	int groupsCount = 1;
	if(blocksCount > MAXBLOCKCOUNT)
	{
		groupsCount = blocksCount / MAXBLOCKCOUNT + 1;
		blocksCount = MAXBLOCKCOUNT;
	}
	
	unsigned long long messageCombination = 0;
	unsigned long long textsCount = 1;
	for(int i = 0; i < MSGLEN; i++)
		textsCount *= signsCount;

	srand(time(NULL));
	hipEventCreate(&timerStart, 0);
	hipEventCreate(&timerStop, 0);
	hipEventRecord(timerStart, 0);

	while(messageCombination < textsCount || option)
	{
		printf("Cracking iteration %lld of %lld\n",messageCombination, textsCount);
		if(!option)
			generatePermutation(messageCombination, signsCount, MSGLEN, code);
		else
			generateRandomPermutation(signsCount, MSGLEN, code);
		convertTextToBitArray(code,MSGLEN,plainBitText);
		messageCombination++;
		if(hipMemcpy(d_plainText, plainBitText, sizeof(short)*MSGBITLEN, hipMemcpyHostToDevice) != hipSuccess)
			ERR("hipMemcpy");
		for(int group = 0; group < groupsCount; group++)
		{
			CrackingDESKernel<<<blocksCount,BLOCKSIZE>>>(d_cipherText, d_plainText, signsCount, threadsCount, group, keyLength, d_result);
			gpuErrchk(hipPeekAtLastError());
			if(hipDeviceSynchronize() != hipSuccess)
				ERR("hipDeviceSynchronize");
			if(hipMemcpy(result, d_result, sizeof(struct result), hipMemcpyDeviceToHost) != hipSuccess)
				ERR("hipMemcpy");
			if(result->isCracked)
				break;
		}
		if(result->isCracked)
		{
			printf("MESSAGE CRACKED\n");
			printf("MSG: ");
			for(int i=0; i < MSGLEN; i++)
				printf("%c",code[i]);
			printf("\n");
			generatePermutation(result->keyNumber, signsCount, MSGLEN, code);
			printf("KEY: ");
			for(int i=0; i < keyLength; i++)
				printf("%c",code[i]);
			printf("\n");
			break;
		}
	}

	if(hipEventRecord(timerStop, 0) != hipSuccess)
		ERR("hipEventRecord");

	if(hipEventSynchronize(timerStop) != hipSuccess)
		ERR("hipEventSynchronize");

	if(hipDeviceSynchronize() != hipSuccess)
		ERR("hipDeviceSynchronize");

	hipEventElapsedTime(&timer, timerStart, timerStop);

	printf("\n");

	printf("TIME = %d s %d ms\n", ((int)timer) / 1000, ((int)timer) % 1000);

	hipEventDestroy(timerStart);
	hipEventDestroy(timerStop);

	if(hipFree(d_cipherText) != hipSuccess)
			ERR("hipFree");
	if(hipFree(d_plainText) != hipSuccess)
			ERR("hipFree");

	delete[] plainText;
	delete[] key;
	delete[] plainBitText;
	delete[] cipherBitText;
	delete[] keyBit;
}